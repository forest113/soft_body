#include <iostream>
#include <math.h>
#include <vector>
#include <unordered_set>
#include <algorithm>
#include <fstream>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "soft_body_parallel.cuh"

extern  Point* G_points_list_d;
extern  Spring* G_springs_list_d;
extern int G_num_points;
extern int G_num_springs;

/* for hash map of edges. */
struct hashFunction
{
	size_t operator()(const vector<int>
		& myVector) const
	{
		std::hash<int> hasher;
		size_t answer = 0;

		for (int i : myVector)
		{
			answer ^= hasher(i) + 0x9e3779b9 +
				(answer << 6) + (answer >> 2);
		}
		return answer;
	}
};

int main(int argc, char** argv) {
	int num_points = 0, num_hex = 0;

	string c;
	ifstream fin;
	/* read the mesh. */
	fin.open("test.hex", ios::in);
	if (!fin) {
		std::cerr << "There was a problem opening the input file!\n";
		exit(1);//exit or do additional error checking
	}
	fin >> num_points;
	cout << num_points << endl;
	/* List of all the vertices. */
	vector<Point> points_list(num_points, Point(0, 0, 0));
	/* List of all the springs. */
	vector<Spring> springs_list;
	/* hashset of edges. */
	unordered_set<vector<int>, hashFunction> edges;
	/* read all points. */
	for (int i = 0; i < num_points; i++) {
		Point p;
		double px, py, pz;
		fin >> px >> py >> pz;
		p = Point(px, py, pz);
		points_list[i] = (p);
	}
	cout << "read" << points_list.size() << "points," << num_points << endl;;

	fin >> num_hex;
	cout << num_hex << endl;
	for (int i = 0; i < num_hex; i++) {
		int p[8] = { 0,0,0,0,0,0,0,0 }, ignore;
		fin >> p[0] >> p[1] >> p[2] >> p[3] >> p[4] >> p[5] >> p[6] >> p[7] >> ignore;
		/* create hashset containing pairs of vertices that form edges, so that springs do not repeat over eachother,
		 * in case of overlapping hexaderals. */
		vector<int> edge(2, 0);
		for (int i = 0; i < 4; i++) {
			/* this step adds edges 0-1, 1-2, 2-3, 3-0. */
			edge[0] = p[i];
			edge[1] = p[(i + 1) % 4];
			sort(edge.begin(), edge.end());
			if (edges.find(edge) == edges.end()) {
				edges.insert(edge);
			}
			/* this step adds edges 4-5, 5-6, 6-7, 7-8. */
			edge[0] = p[i + 4];
			edge[1] = p[((i + 1) % 4) + 4];
			sort(edge.begin(), edge.end());
			if (edges.find(edge) == edges.end()) {
				edges.insert(edge);
			}
			/* this steps adds edges 0-4, 1-5, 2-6, 3-7. */
			edge[0] = p[i];
			edge[1] = p[i + 4];
			sort(edge.begin(), edge.end());
			if (edges.find(edge) == edges.end()) {
				edges.insert(edge);
			}
		}
		edge.clear();
	}
	fin.close();

	cout << "num springs:" << edges.size() << endl;
	springs_list.resize(edges.size(), Spring(NULL, NULL));
	/* create springs from set of edges. */
	unordered_set<vector<int>>::iterator itr;
	int i = 0;
	for (itr = edges.begin();
		itr != edges.end(); itr++)
	{
		Point* p1, * p2;

		vector<int> edge = *itr;
		p1 = &points_list[edge[0] - 1];
		p2 = &points_list[edge[1] - 1];
		
		p1->num_springs += 1;
		p2->num_springs += 1;
		Spring s = Spring(p1, p2);
		s.ind1 = edge[0] - 1;
		s.ind2 = edge[1] - 1;
		springs_list[i] = (s);
		
		i++;
	}

	/* read input. */
	ifstream fin1;
	fin1.open("input.txt", ios::in);
	if (!fin1) {
		std::cerr << "There was a problem opening the input file!\n";
		exit(1);//exit or do additional error checking
	}
	int num_fixed = 0;
	//cout << "Enter number of fixed vertices:" << endl;
	fin1 >> num_fixed;
	for (int i = 0; i < num_fixed; i++) {
		int ind;
		//cout << "Enter " << i << "th" << "fixed point index" << endl;
		fin1 >> ind;
		points_list[ind].is_fixed = true;
	}

	if (num_fixed == 0) {
		//cout << "numspinrgs" << endl;
		for (int i = 0; i < points_list.size(); i++) {
			if (points_list[i].num_springs < 6) {
				points_list[i].is_fixed = true;
			}
		}cout << endl;
	}
	cout << "read something";

	int num_D_n = 0;
	//cout << "Enter number of vertices in D_n" << endl;
	fin1 >> num_D_n;
	for (int i = 0; i < num_D_n; i++) {
		int ind;
		//cout << "enter " << i << "th point in D_n";
		fin1 >> ind;
		Point* p = &points_list[ind];
		p->pos.x = p->pos.x + 0.2;
		//p->is_fixed = false;
	}

	cout << "simulating timesteps" << endl;
	//G_points_list = &points_list;
	hipError_t cudaStatus = hipMalloc(&G_points_list_d, points_list.size() * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMallocManaged failed: %s\n", hipGetErrorString(cudaStatus));
		// handle error condition
	}
	hipMalloc(&G_springs_list_d, springs_list.size() * sizeof(Spring));
	hipMemcpy(G_points_list_d, points_list.data(), points_list.size() * sizeof(Point), hipMemcpyHostToDevice);
	//G_springs_list = &springs_list;
	hipMemcpy(G_springs_list_d, springs_list.data(), springs_list.size() * sizeof(Spring), hipMemcpyHostToDevice);
	G_num_points = points_list.size();
	cout << "num points:" << num_points << endl;
	
	G_num_springs = springs_list.size();
	G_points_list_h = (Point*)malloc(G_num_points * sizeof(Point));
	G_springs_list_h = (Spring*)malloc(G_num_springs * sizeof(Spring));
	fix_springs_pointers_kernel <<<1, 128 >>> (G_springs_list_d, G_points_list_d, G_num_springs);
	test();
	cout<<"yis"<<endl;

	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_SINGLE | GLUT_RGB | GLUT_DEPTH);
	glutInitWindowSize(500, 500);
	glutCreateWindow("OpenGL Window");
	glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
	glutDisplayFunc(display);
	glutTimerFunc(0, update, 0);
	glutMainLoop();

	hipFree(G_points_list_d);
	hipFree(G_springs_list_d);

	/*ofstream fout("sphere.obj");
	//cout << "points:" << endl;
	for (int i = 0; i < points_list.size(); i++) {
		fout << "v " << points_list[i].pos.x << " " << points_list[i].pos.y << " " << points_list[i].pos.z << endl;

	}
	for (itr = edges.begin();
		itr != edges.end(); itr++)
	{

		vector<int> edge = *itr;
		int p1 = edge[0];
		int p2 = edge[1];
		fout << "l " << p1 << " " << p2 << endl;
	}*/





}



